#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>


#include "convolution_common.h"

//#include "convolution.cuh"
#include "convolution_test.cuh"

using namespace std;
extern "C" float convolution(
        float *input,
        float *kernel,
        float *output,

        int width_input,
        int height_input,
        int deep_input,

        int width_kernel,
        int height_kernel,
        int deep_kernel,
        int long_kernel,

        int width_output,
        int height_output,
        int deep_output,


        int stride_x,
        int stride_y,
        int padding_x,
        int padding_y
)
{



    dim3 grid(width_output, height_output, deep_output);
    dim3 thread(width_kernel,height_kernel,deep_kernel);


    hipError_t error;
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    Convolution_kernel<<< grid, thread >>>(
            input,
            kernel,
            output,

            width_input,
            height_input,
            deep_input,

            width_kernel,
            height_kernel,
            deep_kernel,
            long_kernel,

            width_output,
            height_output,
            deep_output,

            stride_x,
            stride_y,
            padding_x,
            padding_y
    );

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return msecTotal;
}